
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

__global__ void kernel(double* a, double* b, double* c, int N) {
	int i, idx = blockDim.x * blockIdx.x + threadIdx.x;	
	int offset = blockDim.x * gridDim.x;				
	for(i = idx; i < N; i += offset) {
		c[i] = (a[i] >= b[i]) ? a[i] : b[i];
    }
}

int main() {
	int N;
	std::cin >> N;
	double* a = new double[N];
    double* b = new double[N];
    double* c = new double[N];
    
	for (int i = 0; i < N; ++i) {
		std::cin >> a[i];
	}
	for (int i = 0; i < N; ++i) {
		std::cin >> b[i];
	}

	double *device_a, *device_b, *device_c;

	hipMalloc((void**) &device_a, N * sizeof(double));
	hipMalloc((void**) &device_b, N * sizeof(double));
	hipMalloc((void**) &device_c, N * sizeof(double));

	hipMemcpy(device_a, a, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(device_b, b, N * sizeof(double), hipMemcpyHostToDevice);

	kernel<<<256, 256>>>(device_a, device_b, device_c, N);

	hipMemcpy(c, device_c, N * sizeof(double), hipMemcpyDeviceToHost);

	std::cout.precision(10);
	std::cout.setf(std::ios::scientific);
	for (int i = 0; i < N; i++) {
		std::cout << c[i] << " ";
	}
	putchar('\n');

	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);
    delete[] a;
    delete[] b;
    delete[] c;

	return 0;
}